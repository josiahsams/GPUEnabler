#include "hip/hip_runtime.h"
#define HIPCUB_STDERR
#include <stdio.h>
#include <iostream>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_reduce.cuh>

using namespace hipcub;
//---------------------------------------------------------------------
// Kernels
//---------------------------------------------------------------------
#define BLOCK_THREADS 64
#define ITEMS_PER_THREAD 16

extern "C"
__global__ void invokeBlockSumKernel( int num,
    long         *d_in,          // Tile of input
    long         *d_out)         // Tile aggregate
{
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;

    // Specialize BlockReduce type for our thread block
    typedef BlockReduce<int, BLOCK_THREADS, BLOCK_REDUCE_WARP_REDUCTIONS> BlockReduceT;
    // Shared memory
    __shared__ typename BlockReduceT::TempStorage temp_storage;
    // Per-thread tile data
    int data[ITEMS_PER_THREAD];
    LoadDirectStriped<BLOCK_THREADS>(ix, d_in, data);

    // Compute sum
    long aggregate = BlockReduceT(temp_storage).Sum(data);
    if (ix == 0)
    {
       *d_out = aggregate;
    }

}

